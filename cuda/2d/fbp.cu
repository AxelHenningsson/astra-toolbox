#include "hip/hip_runtime.h"
/*
-----------------------------------------------------------------------
Copyright: 2010-2018, imec Vision Lab, University of Antwerp
           2014-2018, CWI, Amsterdam

Contact: astra@astra-toolbox.com
Website: http://www.astra-toolbox.com/

This file is part of the ASTRA Toolbox.


The ASTRA Toolbox is free software: you can redistribute it and/or modify
it under the terms of the GNU General Public License as published by
the Free Software Foundation, either version 3 of the License, or
(at your option) any later version.

The ASTRA Toolbox is distributed in the hope that it will be useful,
but WITHOUT ANY WARRANTY; without even the implied warranty of
MERCHANTABILITY or FITNESS FOR A PARTICULAR PURPOSE. See the
GNU General Public License for more details.

You should have received a copy of the GNU General Public License
along with the ASTRA Toolbox. If not, see <http://www.gnu.org/licenses/>.

-----------------------------------------------------------------------
$Id$
*/

#include "fbp.h"
#include "fft.h"
#include "par_bp.h"
#include "fan_bp.h"
#include "util.h"

// For fan-beam preweighting
#include "../3d/fdk.h"

#include "astra/Logging.h"

#include <hip/hip_runtime.h>

namespace astraCUDA {



static int calcNextPowerOfTwo(int n)
{
	int x = 1;
	while (x < n)
		x *= 2;

	return x;
}

// static
int FBP::calcFourierFilterSize(int _iDetectorCount)
{
	int iFFTRealDetCount = calcNextPowerOfTwo(2 * _iDetectorCount);
	int iFreqBinCount = calcFFTFourierSize(iFFTRealDetCount);

	// CHECKME: Matlab makes this at least 64. Do we also need to?
	return iFreqBinCount;
}




FBP::FBP() : ReconAlgo()
{
	D_filter = 0;

}

FBP::~FBP()
{
	reset();
}

void FBP::reset()
{
	if (D_filter) {
		freeComplexOnDevice((hipfftComplex *)D_filter);
		D_filter = 0;
	}
}

bool FBP::init()
{
	return true;
}

bool FBP::setFilter(astra::E_FBPFILTER _eFilter, const float * _pfHostFilter /* = NULL */, int _iFilterWidth /* = 0 */, float _fD /* = 1.0f */, float _fFilterParameter /* = -1.0f */)
{
	if (D_filter)
	{
		freeComplexOnDevice((hipfftComplex*)D_filter);
		D_filter = 0;
	}

	if (_eFilter == astra::FILTER_NONE)
		return true; // leave D_filter set to 0


	int iFFTRealDetCount = calcNextPowerOfTwo(2 * dims.iProjDets);
	int iFreqBinCount = calcFFTFourierSize(iFFTRealDetCount);

	hipfftComplex * pHostFilter = new hipfftComplex[dims.iProjAngles * iFreqBinCount];
	memset(pHostFilter, 0, sizeof(hipfftComplex) * dims.iProjAngles * iFreqBinCount);

	allocateComplexOnDevice(dims.iProjAngles, iFreqBinCount, (hipfftComplex**)&D_filter);

	switch(_eFilter)
	{
		case astra::FILTER_NONE:
			// handled above
			break;
		case astra::FILTER_RAMLAK:
		case astra::FILTER_SHEPPLOGAN:
		case astra::FILTER_COSINE:
		case astra::FILTER_HAMMING:
		case astra::FILTER_HANN:
		case astra::FILTER_TUKEY:
		case astra::FILTER_LANCZOS:
		case astra::FILTER_TRIANGULAR:
		case astra::FILTER_GAUSSIAN:
		case astra::FILTER_BARTLETTHANN:
		case astra::FILTER_BLACKMAN:
		case astra::FILTER_NUTTALL:
		case astra::FILTER_BLACKMANHARRIS:
		case astra::FILTER_BLACKMANNUTTALL:
		case astra::FILTER_FLATTOP:
		case astra::FILTER_PARZEN:
		{
			genFilter(_eFilter, _fD, dims.iProjAngles, pHostFilter, iFFTRealDetCount, iFreqBinCount, _fFilterParameter);
			uploadComplexArrayToDevice(dims.iProjAngles, iFreqBinCount, pHostFilter, (hipfftComplex*)D_filter);

			break;
		}
		case astra::FILTER_PROJECTION:
		{
			// make sure the offered filter has the correct size
			assert(_iFilterWidth == iFreqBinCount);

			for(int iFreqBinIndex = 0; iFreqBinIndex < iFreqBinCount; iFreqBinIndex++)
			{
				float fValue = _pfHostFilter[iFreqBinIndex];

				for(int iProjectionIndex = 0; iProjectionIndex < (int)dims.iProjAngles; iProjectionIndex++)
				{
					pHostFilter[iFreqBinIndex + iProjectionIndex * iFreqBinCount].x = fValue;
					pHostFilter[iFreqBinIndex + iProjectionIndex * iFreqBinCount].y = 0.0f;
				}
			}
			uploadComplexArrayToDevice(dims.iProjAngles, iFreqBinCount, pHostFilter, (hipfftComplex*)D_filter);
			break;
		}
		case astra::FILTER_SINOGRAM:
		{
			// make sure the offered filter has the correct size
			assert(_iFilterWidth == iFreqBinCount);

			for(int iFreqBinIndex = 0; iFreqBinIndex < iFreqBinCount; iFreqBinIndex++)
			{
				for(int iProjectionIndex = 0; iProjectionIndex < (int)dims.iProjAngles; iProjectionIndex++)
				{
					float fValue = _pfHostFilter[iFreqBinIndex + iProjectionIndex * _iFilterWidth];

					pHostFilter[iFreqBinIndex + iProjectionIndex * iFreqBinCount].x = fValue;
					pHostFilter[iFreqBinIndex + iProjectionIndex * iFreqBinCount].y = 0.0f;
				}
			}
			uploadComplexArrayToDevice(dims.iProjAngles, iFreqBinCount, pHostFilter, (hipfftComplex*)D_filter);
			break;
		}
		case astra::FILTER_RPROJECTION:
		{
			int iProjectionCount = dims.iProjAngles;
			int iRealFilterElementCount = iProjectionCount * iFFTRealDetCount;
			float * pfHostRealFilter = new float[iRealFilterElementCount];
			memset(pfHostRealFilter, 0, sizeof(float) * iRealFilterElementCount);

			int iUsedFilterWidth = min(_iFilterWidth, iFFTRealDetCount);
			int iStartFilterIndex = (_iFilterWidth - iUsedFilterWidth) / 2;
			int iMaxFilterIndex = iStartFilterIndex + iUsedFilterWidth;

			int iFilterShiftSize = _iFilterWidth / 2;

			for(int iDetectorIndex = iStartFilterIndex; iDetectorIndex < iMaxFilterIndex; iDetectorIndex++)
			{
				int iFFTInFilterIndex = (iDetectorIndex + iFFTRealDetCount - iFilterShiftSize) % iFFTRealDetCount;
				float fValue = _pfHostFilter[iDetectorIndex];

				for(int iProjectionIndex = 0; iProjectionIndex < (int)dims.iProjAngles; iProjectionIndex++)
				{
					pfHostRealFilter[iFFTInFilterIndex + iProjectionIndex * iFFTRealDetCount] = fValue;
				}
			}

			float* pfDevRealFilter = NULL;
			hipMalloc((void **)&pfDevRealFilter, sizeof(float) * iRealFilterElementCount); // TODO: check for errors
			hipMemcpy(pfDevRealFilter, pfHostRealFilter, sizeof(float) * iRealFilterElementCount, hipMemcpyHostToDevice);
			delete[] pfHostRealFilter;

			runCudaFFT(iProjectionCount, pfDevRealFilter, iFFTRealDetCount, iFFTRealDetCount, iFFTRealDetCount, iFreqBinCount, (hipfftComplex*)D_filter);

			hipFree(pfDevRealFilter);

			break;
		}
		case astra::FILTER_RSINOGRAM:
		{
			int iProjectionCount = dims.iProjAngles;
			int iRealFilterElementCount = iProjectionCount * iFFTRealDetCount;
			float* pfHostRealFilter = new float[iRealFilterElementCount];
			memset(pfHostRealFilter, 0, sizeof(float) * iRealFilterElementCount);

			int iUsedFilterWidth = min(_iFilterWidth, iFFTRealDetCount);
			int iStartFilterIndex = (_iFilterWidth - iUsedFilterWidth) / 2;
			int iMaxFilterIndex = iStartFilterIndex + iUsedFilterWidth;

			int iFilterShiftSize = _iFilterWidth / 2;

			for(int iDetectorIndex = iStartFilterIndex; iDetectorIndex < iMaxFilterIndex; iDetectorIndex++)
			{
				int iFFTInFilterIndex = (iDetectorIndex + iFFTRealDetCount - iFilterShiftSize) % iFFTRealDetCount;

				for(int iProjectionIndex = 0; iProjectionIndex < (int)dims.iProjAngles; iProjectionIndex++)
				{
					float fValue = _pfHostFilter[iDetectorIndex + iProjectionIndex * _iFilterWidth];
					pfHostRealFilter[iFFTInFilterIndex + iProjectionIndex * iFFTRealDetCount] = fValue;
				}
			}

			float* pfDevRealFilter = NULL;
			hipMalloc((void **)&pfDevRealFilter, sizeof(float) * iRealFilterElementCount); // TODO: check for errors
			hipMemcpy(pfDevRealFilter, pfHostRealFilter, sizeof(float) * iRealFilterElementCount, hipMemcpyHostToDevice);
			delete[] pfHostRealFilter;

			runCudaFFT(iProjectionCount, pfDevRealFilter, iFFTRealDetCount, iFFTRealDetCount, iFFTRealDetCount, iFreqBinCount, (hipfftComplex*)D_filter);

			hipFree(pfDevRealFilter);

			break;
		}
		default:
		{
			ASTRA_ERROR("FBP::setFilter: Unknown filter type requested");
			delete [] pHostFilter;
			return false;
		}
	}

	delete [] pHostFilter;

	return true;
}

bool FBP::iterate(unsigned int iterations)
{
	zeroVolumeData(D_volumeData, volumePitch, dims);

	bool ok = false;

	if (fanProjs) {
		// Call FDK_PreWeight to handle fan beam geometry. We treat
		// this as a cone beam setup of a single slice:

		// TODO: TOffsets affects this preweighting...

		// TODO: We take the fan parameters from the last projection here
		// without checking if they're the same in all projections

		float *pfAngles = new float[dims.iProjAngles];

		float fOriginSource, fOriginDetector, fDetSize, fOffset;
		for (unsigned int i = 0; i < dims.iProjAngles; ++i) {
			bool ok = astra::getFanParameters(fanProjs[i], dims.iProjDets,
			                                  pfAngles[i],
			                                  fOriginSource, fOriginDetector,
			                                  fDetSize, fOffset);
			if (!ok) {
				ASTRA_ERROR("FBP_CUDA: Failed to extract circular fan beam parameters from fan beam geometry");
				return false;
			}
		}

		// We create a fake hipPitchedPtr
		hipPitchedPtr tmp;
		tmp.ptr = D_sinoData;
		tmp.pitch = sinoPitch * sizeof(float);
		tmp.xsize = dims.iProjDets;
		tmp.ysize = dims.iProjAngles;
		// and a fake Dimensions3D
		astraCUDA3d::SDimensions3D dims3d;
		dims3d.iVolX = dims.iVolWidth;
		dims3d.iVolY = dims.iVolHeight;
		dims3d.iVolZ = 1;
		dims3d.iProjAngles = dims.iProjAngles;
		dims3d.iProjU = dims.iProjDets;
		dims3d.iProjV = 1;

		astraCUDA3d::FDK_PreWeight(tmp, fOriginSource,
		              fOriginDetector, 0.0f,
		              fDetSize, 1.0f,
		              m_bShortScan, dims3d, pfAngles);
	} else {
		// TODO: How should different detector pixel size in different
		// projections be handled?
	}

	if (D_filter) {

		int iFFTRealDetCount = calcNextPowerOfTwo(2 * dims.iProjDets);
		int iFFTFourDetCount = calcFFTFourierSize(iFFTRealDetCount);

		hipfftComplex * pDevComplexSinogram = NULL;

		allocateComplexOnDevice(dims.iProjAngles, iFFTFourDetCount, &pDevComplexSinogram);

		runCudaFFT(dims.iProjAngles, D_sinoData, sinoPitch, dims.iProjDets, iFFTRealDetCount, iFFTFourDetCount, pDevComplexSinogram);

		applyFilter(dims.iProjAngles, iFFTFourDetCount, pDevComplexSinogram, (hipfftComplex*)D_filter);

		runCudaIFFT(dims.iProjAngles, pDevComplexSinogram, D_sinoData, sinoPitch, dims.iProjDets, iFFTRealDetCount, iFFTFourDetCount);

		freeComplexOnDevice(pDevComplexSinogram);

	}

	float fOutputScale = (M_PI / 2.0f) / (float)dims.iProjAngles;

	if (fanProjs) {
		ok = FanBP_FBPWeighted(D_volumeData, volumePitch, D_sinoData, sinoPitch, dims, fanProjs, fOutputScale);

	} else {
		ok = BP(D_volumeData, volumePitch, D_sinoData, sinoPitch, dims, parProjs, fOutputScale);
	}
	if(!ok)
	{
		return false;
	}

	return true;
}


}
