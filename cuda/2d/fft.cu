#include "hip/hip_runtime.h"
/*
-----------------------------------------------------------------------
Copyright: 2010-2022, imec Vision Lab, University of Antwerp
           2014-2022, CWI, Amsterdam

Contact: astra@astra-toolbox.com
Website: http://www.astra-toolbox.com/

This file is part of the ASTRA Toolbox.


The ASTRA Toolbox is free software: you can redistribute it and/or modify
it under the terms of the GNU General Public License as published by
the Free Software Foundation, either version 3 of the License, or
(at your option) any later version.

The ASTRA Toolbox is distributed in the hope that it will be useful,
but WITHOUT ANY WARRANTY; without even the implied warranty of
MERCHANTABILITY or FITNESS FOR A PARTICULAR PURPOSE. See the
GNU General Public License for more details.

You should have received a copy of the GNU General Public License
along with the ASTRA Toolbox. If not, see <http://www.gnu.org/licenses/>.

-----------------------------------------------------------------------
*/

#include "astra/cuda/2d/fft.h"
#include "astra/cuda/2d/util.h"

#include "astra/Logging.h"
#include "astra/Fourier.h"

#include <hipfft/hipfft.h>
#include <hip/hip_runtime.h>


using namespace astra;

namespace astraCUDA {

bool checkCufft(hipfftResult err, const char *msg)
{
	if (err != HIPFFT_SUCCESS) {
		ASTRA_ERROR("%s: CUFFT error %d.", msg, err);
		return false;
	} else {
		return true;
	}
}

__global__ static void applyFilter_kernel(int _iProjectionCount,
                                          int _iFreqBinCount,
                                          hipfftComplex * _pSinogram,
                                          hipfftComplex * _pFilter)
{
	int iIndex = threadIdx.x + blockIdx.x * blockDim.x;
	int iProjectionIndex = iIndex / _iFreqBinCount;

	if(iProjectionIndex >= _iProjectionCount)
	{
		return;
	}

	float fA = _pSinogram[iIndex].x;
	float fB = _pSinogram[iIndex].y;
	float fC = _pFilter[iIndex].x;
	float fD = _pFilter[iIndex].y;

	_pSinogram[iIndex].x = fA * fC - fB * fD;
	_pSinogram[iIndex].y = fA * fD + fC * fB;
}

__global__ static void rescaleInverseFourier_kernel(int _iProjectionCount,
                                                    int _iDetectorCount,
                                                    float* _pfInFourierOutput)
{
	int iIndex = threadIdx.x + blockIdx.x * blockDim.x;
	int iProjectionIndex = iIndex / _iDetectorCount;
	int iDetectorIndex = iIndex % _iDetectorCount;

	if(iProjectionIndex >= _iProjectionCount)
	{
		return;
	}

	_pfInFourierOutput[iProjectionIndex * _iDetectorCount + iDetectorIndex] /= (float)_iDetectorCount;
}

bool rescaleInverseFourier(int _iProjectionCount, int _iDetectorCount,
                           float * _pfInFourierOutput,
                           std::optional<hipStream_t> _stream)
{
	StreamHelper stream(_stream);
	if (!stream)
		return false;

	const int iBlockSize = 256;
	int iElementCount = _iProjectionCount * _iDetectorCount;
	int iBlockCount = (iElementCount + iBlockSize - 1) / iBlockSize;

	rescaleInverseFourier_kernel<<< iBlockCount, iBlockSize, 0, stream() >>>(_iProjectionCount,
	                                                            _iDetectorCount,
	                                                            _pfInFourierOutput);

	return stream.syncIfSync("rescaleInverseFourier");
}

bool applyFilter(int _iProjectionCount, int _iFreqBinCount,
                 hipfftComplex * _pSinogram, hipfftComplex * _pFilter,
                 std::optional<hipStream_t> _stream)
{
	StreamHelper stream(_stream);
	if (!stream)
		return false;

	const int iBlockSize = 256;
	int iElementCount = _iProjectionCount * _iFreqBinCount;
	int iBlockCount = (iElementCount + iBlockSize - 1) / iBlockSize;

	applyFilter_kernel<<< iBlockCount, iBlockSize, 0, stream() >>>(_iProjectionCount,
	                                                  _iFreqBinCount,
	                                                  _pSinogram, _pFilter);

	return stream.syncIfSync("applyFilter");
}

static bool invokeCudaFFT(int _iProjectionCount, int _iDetectorCount,
                          const float * _pfDevSource,
                          hipfftComplex * _pDevTargetComplex,
                          hipStream_t stream)
{
	hipfftHandle plan;

	if (!checkCufft(hipfftPlan1d(&plan, _iDetectorCount, HIPFFT_R2C, _iProjectionCount), "invokeCudaFFT plan")) {
		return false;
	}

	if (!checkCufft(hipfftSetStream(plan, stream), "invokeCudaFFT plan stream")) {
		hipfftDestroy(plan);
		return false;
	}

	if (!checkCufft(hipfftExecR2C(plan, (hipfftReal *)_pfDevSource, _pDevTargetComplex), "invokeCudaFFT exec")) {
		hipfftDestroy(plan);
		return false;
	}

	if (!checkCuda(hipStreamSynchronize(stream), "invokeCudaFFT sync")) {
		hipfftDestroy(plan);
		return false;
	}

	hipfftDestroy(plan);
	return true;
}

static bool invokeCudaIFFT(int _iProjectionCount, int _iDetectorCount,
                           const hipfftComplex * _pDevSourceComplex,
                           float * _pfDevTarget,
                           hipStream_t stream)
{
	hipfftHandle plan;

	if (!checkCufft(hipfftPlan1d(&plan, _iDetectorCount, HIPFFT_C2R, _iProjectionCount), "invokeCudaIFFT plan")) {
		return false;
	}

	if (!checkCufft(hipfftSetStream(plan, stream), "invokeCudaIFFT plan stream")) {
		hipfftDestroy(plan);
		return false;
	}

	// Getting rid of the const qualifier is due to cufft API issue?
	if (!checkCufft(hipfftExecC2R(plan, (hipfftComplex *)_pDevSourceComplex,
	                      (hipfftReal *)_pfDevTarget), "invokeCudaIFFT exec"))
	{
		hipfftDestroy(plan);
		return false;
	}

	if (!checkCuda(hipStreamSynchronize(stream), "invokeCudaIFFT sync")) {
		hipfftDestroy(plan);
		return false;
	}

	hipfftDestroy(plan);
	return true;
}

bool allocateComplexOnDevice(int _iProjectionCount, int _iDetectorCount,
                             hipfftComplex ** _ppDevComplex)
{
	size_t bufferSize = sizeof(hipfftComplex) * _iProjectionCount * _iDetectorCount;
	return checkCuda(hipMalloc((void **)_ppDevComplex, bufferSize), "fft allocateComplexOnDevice");
}

bool freeComplexOnDevice(hipfftComplex * _pDevComplex)
{
	return checkCuda(hipFree(_pDevComplex), "fft freeComplexOnDevice");
}

bool uploadComplexArrayToDevice(int _iProjectionCount, int _iDetectorCount,
                                hipfftComplex * _pHostComplexSource,
                                hipfftComplex * _pDevComplexTarget,
                                std::optional<hipStream_t> _stream)
{
	StreamHelper stream(_stream);
	if (!stream)
		return false;

	size_t memSize = sizeof(hipfftComplex) * _iProjectionCount * _iDetectorCount;
	bool ok = checkCuda(hipMemcpyAsync(_pDevComplexTarget, _pHostComplexSource, memSize, hipMemcpyHostToDevice, stream()), "fft uploadComplexArrayToDevice");

	ok &= stream.syncIfSync("fft uploadComplexArrayToDevice");
	return ok;
}

bool runCudaFFT(int _iProjectionCount,
                const float * D_pfSource, int _iSourcePitch,
                int _iProjDets, int _iPaddedSize,
                hipfftComplex * D_pcTarget,
                std::optional<hipStream_t> _stream)
{
	StreamHelper stream(_stream);
	if (!stream)
		return false;

	float * D_pfPaddedSource = NULL;
	size_t bufferMemSize = sizeof(float) * _iProjectionCount * _iPaddedSize;

	if (!checkCuda(hipMalloc((void **)&D_pfPaddedSource, bufferMemSize), "runCudaFFT malloc")) {
		return false;
	}
	if (!checkCuda(hipMemsetAsync(D_pfPaddedSource, 0, bufferMemSize, stream()), "runCudaFFT memset")) {
		hipFree(D_pfPaddedSource);
		return false;
	}

	// pitched memcpy 2D to handle both source pitch and target padding
	if (!checkCuda(hipMemcpy2DAsync(D_pfPaddedSource, _iPaddedSize*sizeof(float), D_pfSource, _iSourcePitch*sizeof(float), _iProjDets*sizeof(float), _iProjectionCount, hipMemcpyDeviceToDevice, stream()), "runCudaFFT memcpy")) {
		hipFree(D_pfPaddedSource);
		return false;
	}

	if (!invokeCudaFFT(_iProjectionCount, _iPaddedSize, D_pfPaddedSource, D_pcTarget, stream())) {
		hipFree(D_pfPaddedSource);
		return false;
	}

	if (!stream.sync("runCudaFFT sync")) {
		hipFree(D_pfPaddedSource);
		return false;
	}

	hipFree(D_pfPaddedSource);
	return true;
}

bool runCudaIFFT(int _iProjectionCount, const hipfftComplex *D_pcSource,
                 float * D_pfTarget, int _iTargetPitch,
                 int _iProjDets, int _iPaddedSize,
                 std::optional<hipStream_t> _stream)
{
	StreamHelper stream(_stream);
	if (!stream)
		return false;

	float * D_pfPaddedTarget = NULL;
	size_t bufferMemSize = sizeof(float) * _iProjectionCount * _iPaddedSize;

	if (!checkCuda(hipMalloc((void **)&D_pfPaddedTarget, bufferMemSize), "runCudaIFFT malloc")) {
		return false;
	}

	if (!invokeCudaIFFT(_iProjectionCount, _iPaddedSize,
	                    D_pcSource, D_pfPaddedTarget, stream()))
	{
		hipFree(D_pfPaddedTarget);
		return false;
	}

	rescaleInverseFourier(_iProjectionCount, _iPaddedSize,
	                      D_pfPaddedTarget, stream());

	if (!checkCuda(hipMemsetAsync(D_pfTarget, 0, sizeof(float) * _iProjectionCount * _iTargetPitch, stream()), "runCudaIFFT memset")) {
		hipFree(D_pfPaddedTarget);
		return false;
	}

	// pitched memcpy 2D to handle both source padding and target pitch
	if (!checkCuda(hipMemcpy2DAsync(D_pfTarget, _iTargetPitch*sizeof(float), D_pfPaddedTarget, _iPaddedSize*sizeof(float), _iProjDets*sizeof(float), _iProjectionCount, hipMemcpyDeviceToDevice, stream()), "runCudaIFFT memcpy")) {
		hipFree(D_pfPaddedTarget);
		return false;
	}

	if (!stream.sync("runCudaIFFT sync")) {
		hipFree(D_pfPaddedTarget);
		return false;
	}

	hipFree(D_pfPaddedTarget);
	return true;
}

void genIdenFilter(int _iProjectionCount, hipfftComplex * _pFilter,
                   int _iFFTRealDetectorCount, int _iFFTFourierDetectorCount)
{
	for(int iProjectionIndex = 0; iProjectionIndex < _iProjectionCount; iProjectionIndex++)
	{
		for(int iDetectorIndex = 0; iDetectorIndex < _iFFTFourierDetectorCount; iDetectorIndex++)
		{
			int iIndex = iDetectorIndex + iProjectionIndex * _iFFTFourierDetectorCount;
			_pFilter[iIndex].x = 1.0f;
			_pFilter[iIndex].y = 0.0f;
		}
	}
}

void genCuFFTFilter(const SFilterConfig &_cfg, int _iProjectionCount,
               hipfftComplex * _pFilter, int _iFFTRealDetectorCount,
               int _iFFTFourierDetectorCount)
{
	float * pfFilt = astra::genFilter(_cfg,
	                                  _iFFTRealDetectorCount,
	                                  _iFFTFourierDetectorCount);

	for(int iDetectorIndex = 0; iDetectorIndex < _iFFTFourierDetectorCount; iDetectorIndex++)
	{
		float fFilterValue = pfFilt[iDetectorIndex];

		for(int iProjectionIndex = 0; iProjectionIndex < _iProjectionCount; iProjectionIndex++)
		{
			int iIndex = iDetectorIndex + iProjectionIndex * _iFFTFourierDetectorCount;
			_pFilter[iIndex].x = fFilterValue;
			_pFilter[iIndex].y = 0.0f;
		}
	}

	delete[] pfFilt;
}


}
